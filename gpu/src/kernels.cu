#include "hip/hip_runtime.h"
/** GPU Kernels.
 *
 * @file
 * @author Daniel Krebs <github@daniel-krebs.net>
 * @copyright 2017-2022, Institute for Automation of Complex Power Systems, EONERC
 * @license GNU General Public License (version 3)
 *
 * VILLASfpga
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 *********************************************************************************/

#include <stdio.h>

#include <villas/gpu.hpp>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "kernels.hpp"

using namespace villas::gpu;


__global__ void
kernel_mailbox(volatile uint32_t *mailbox, volatile uint32_t* counter)
{
	printf("[gpu] hello!\n");
	printf("[gpu] mailbox: %p\n", mailbox);

	printf("[kernel] started\n");

	while (1) {
		if (*mailbox == 1) {
			*mailbox = 0;
			printf("[gpu] counter = %d\n", *counter);
			break;
		}
	}

	printf("[gpu] quit\n");
}

__global__ void
kernel_memcpy(volatile uint8_t* dst, volatile uint8_t* src, size_t length)
{
	while (length > 0) {
		*dst++ = *src++;
		length--;
	}
}
